#include "hip/hip_runtime.h"
// NOTE: CUSP's own IO library is so poorly written and has really really no efficiency.
// Compile with  nvcc -I./ -arch=sm_20 -O2 test/mult.cu
#include <cusp/multiply.h>
#include <cusp/io/matrix_market.h>
#include <cusp/coo_matrix.h>
#include <cusp/csr_matrix.h>
#include <cusp/print.h>
#include <vector>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
const int N = 61578170;
const int M = 345439900;
const float DAMPINGFACTOR = 0.85;
cusp::coo_matrix<int, float, cusp::host_memory> B(N,N,M);
const int niter = 4;
const char mtxFile[] = "/media/tmp/graphchi/data/test3";

void FIXLINE(char *s){
	int l = (int)strlen(s)-1;
	if(s[l] == '\n')s[l]=0;
}

void my_read_matrix(){
	int cnt=0;
	FILE *fp = fopen(mtxFile,"r");
	char s[1024];
	double diff;
	time_t time0,time1;
	time(&time0);
	while(fgets(s, 1024, fp) != NULL){
		FIXLINE(s);
		char del[] = "\t ";
		if(s[0]=='#' || s[0] == '%') continue;
		char *t;
		int a,b;
		float c;
		t=strtok(s,del);
		a=atoi(t);
		t=strtok(NULL,del);
		b=atoi(t);
		t=strtok(NULL,del);
		c=atof(t);
		B.row_indices[cnt] = a;
		B.column_indices[cnt] = b;
		B.values[cnt] = c;
		cnt++;
	}
	printf("\n");
	time(&time1);
	diff = difftime(time1, time0);
	printf("Reading %d lines takes %.3f\n", cnt, diff);
}

int main(void)
{
	time_t t0,t1;
	double diff;
	hipSetDevice(0);

	my_read_matrix();

	cusp::coo_matrix<int, float, cusp::device_memory> A(B);
	cusp::array1d<float, cusp::device_memory> x(A.num_cols, 1);
	cusp::array1d<float, cusp::device_memory> y(A.num_rows, 1);
	cusp::array1d<float, cusp::device_memory> z(A.num_rows, (1-DAMPINGFACTOR)/N);

	// for each iteration,  y<-A*x, x<-z, x<-D*y + x
	for(int i=0;i<niter;i++){
		clock_t tc;
		tc = clock();
		cusp::multiply(A, x, y);
		x=z;
		cusp::blas::axpy(y,x,DAMPINGFACTOR);
		diff = ((float)clock() - tc)/CLOCKS_PER_SEC;
		printf("iteration takes %.5f\n",diff);
	}
	time(&t0);
	cusp::array1d<float, cusp::host_memory> yy(x);
	time(&t1);
	diff = difftime(t1,t0);
	printf("copying host takes %.3f\n",diff);
	time(&t0);

	std::vector<float> yyy(yy.begin(), yy.end());
	for(int i=0;i<30;i++)
		printf("%.8f\n",yyy[i]);
	return 0;
}

