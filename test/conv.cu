#include "hip/hip_runtime.h"
#include <cusp/io/matrix_market.h>
#include <cusp/coo_matrix.h>
#include <cusp/print.h>
#include <algorithm>
#include <vector>
#include <set>
#include <utility>
#include <ctime>
const int N = 23026589;
const int M = 324874844; 
const char outFile[] = "test3";
const float probability = 0.25;
cusp::coo_matrix<int,float,cusp::host_memory> A(N,N,M);
int maxV=0,lines=0;
std::vector <std::pair<int,int> > invData;
std::vector <std::pair<int, std::pair<int, float> > > outData;
std::map<int, int> mapped;
int outDegree[61578414];

void FIXLINE(char *s){
	int l = (int)strlen(s)-1;
	if(s[l] == '\n')s[l]=0;
}

void addVertex(int a){
	if(mapped.find(a) == mapped.end()){
		mapped[a] = mapped.size();
		maxV = max(maxV, a);
	}
}

void readConv(float prob){
	time_t time0,time1;
	double diff;
	char s[1024];
	FILE *fp = fopen("twitter_rv.net","r");
	int originalMaxV = 0;
	memset(outDegree, 0, sizeof(outDegree[0])*61578414);
	int curline = 0;
	time(&time0);
	srand(time(NULL));
	while(fgets(s, 1024, fp) != NULL ){
		FIXLINE(s);
		char del[] = "\t ";
		if(s[0]=='#' || s[0] == '%') continue;
		double tmp = rand()/(double)RAND_MAX;
		if(tmp>prob)continue;
		char *t;
		int a,b;
		t=strtok(s,del);
		a=atoi(t);
		t=strtok(NULL,del);
		b=atoi(t);
		originalMaxV = max(originalMaxV, max(a,b));
		invData.push_back(std::make_pair(b,a));
		curline++;
	}
	time(&time1);
	diff = difftime(time1, time0);
	printf("here %d lines reading takes %.3f\n",curline,diff);
	sort(invData.begin(),invData.end());
	int n=invData.size();
	time(&time0);
	for(int i=0;i<n;i++){
		int v=invData[i].first,u=invData[i].second;
		outDegree[u]++;
	}
	time(&time1);
	diff = difftime(time1, time0);
	printf("here counting outdegrees takes %.3f\nstart removing redundant vertices and renaming the rest\n", diff);
	time(&time0);
	for(int i=0;i<n;i++){
		int a = invData[i].second, b = invData[i].first;
		if(outDegree[a] == 0 || outDegree[b] == 0)
			continue;
		lines++;
		addVertex(a);
		addVertex(b);
	}
	time(&time1);
	diff = difftime(time1, time0);
	printf("unique renaming takes %.3f\n",diff);
	printf("%d,%d\n", maxV, lines);
	time(&time0);
	FILE *fout = fopen(outFile,"w");
	for(int i=0;i<n;i++){
		int a = invData[i].second, b = invData[i].first;
		//if(outDegreeTemp[a] == 0||outDegreeTemp[b] ==0)continue;
		if(outDegree[a] == 0 || outDegree[b] == 0)continue;
		outData.push_back(std::make_pair(mapped[b], std::make_pair(mapped[a], 1.0/outDegree[b])));
	}
	sort(outData.begin(), outData.end());
	int m = outData.size();
	for(int i=0;i<m;i++){
		int a = outData[i].first, b = outData[i].second.first;
		float c = outData[i].second.second;
		fprintf(fout,"%d %d %.8f\n", a, b, c);
	}
	fclose(fout);
	fclose(fp);
}

void writeConv(){
	int cnt=0;
	FILE *fp = fopen(outFile,"r");
	char s[1024];
	double diff;
	time_t time0,time1;
	time(&time0);
	while(fgets(s, 1024, fp) != NULL){
		FIXLINE(s);
		char del[] = "\t ";
		if(s[0]=='#' || s[0] == '%') continue;
		char *t;
		int a,b;
		float c;
		t=strtok(s,del);
		a=atoi(t);
		t=strtok(NULL,del);
		b=atoi(t);
		t=strtok(NULL,del);
		c=atof(t);
		A.row_indices[cnt] = a;
		A.column_indices[cnt] = b;
		A.values[cnt] = c;
		cnt++;
		printf("\r%d",cnt);
		fflush(stdin);
	}
	printf("\n");
	time(&time1);
	diff = difftime(time1, time0);
	printf("here construction takes %.3f\n", diff);
	time(&time0);
	cusp::io::write_matrix_market_file(A, "A.mtx");
	time(&time1);
	diff = difftime(time1, time0);
	printf("here output takes %.3f\n", diff);
	fclose(fp);
}

int main()
{
	readConv(probability);
	//writeConv();
	return 0;
}

